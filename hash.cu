#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include "hash.cuh"

__device__ inline unsigned int index2(unsigned int i, unsigned int j){
    return (unsigned int) (i*BW+j);
}

__device__ inline uint32_t ROR2(uint32_t x, int y){
    int y_mod = ((y & 0x1F) + 32) & 0x1F;
    return ROR32(x, y_mod);
}

//note: output must be 32+1 chars (+1 for termination of string)
void Hash(char* input, char* output)
{
    
    //uint32_t a[MS];
    //uint32_t b[BL*BW];
    //uint32_t in[BW];
    //uint32_t out[2];
    //unsigned int d_i;
    
    // copy host memory to device //JR
    hipMemcpy(d_input, input, SIZE_INPUT , hipMemcpyHostToDevice);

    //init with zeros
    hipMemset(d_a,0,SIZE_A);
    hipMemset(d_b,0,SIZE_B);
    hipMemset(d_p,0,sizeof(uint32_t));
    
    uint32_t inputSize= (uint32_t)strlen(input);

    unsigned int p = 0;
    while(p+sizeof(uint32_t)*BW <=inputSize) {
        //inLoop<<<1,1>>>(d_in, d_input, d_p);
        p += sizeof(uint32_t)*BW;
        //InputFunction<<<1,1>>>(d_in,d_a,d_b);
        RoundFunction<<<1,1>>>(d_a,d_b);
    }
#if 0    
    //*debug = a[0];//debug
  
    //padding
    //char* last_block = (char*) calloc(BW+1, sizeof(uint32_t));
    char last_block[(BW+1)*sizeof(uint32_t)];
    for(unsigned int i=0; i<(BW+1)*sizeof(uint32_t); i++)
        last_block[i] = 0;
    
    for(uint32_t i=0;i<inputSize-p;i++)
        last_block[i]=input[p+i];
    last_block[inputSize-p]=(char) 0x01;
    
    d_p = 0;
    inLoop(in,last_block,&d_p);
    InputFunction(in,a,b);
    RoundFunction(a,b);
    (*debug)++;
 
   //do some iterations without new input
    for(uint32_t i=0; i<BI; i++){
        RoundFunction(a,b);
        (*debug)++;
    }
    //*debug = a[0];//debug
    
    //collect 32 output characters
    d_i = 0;
    for(uint32_t i=0;i<32/(2*sizeof(uint32_t));i++){
        RoundFunction(a,b);
        (*debug)++;
        OutputFunction(out,a);
        outLoop(out, output, &d_i);
    }
    output[32]='\0';
#endif
}
    
__global__ void RoundFunction(uint32_t* a, uint32_t* b)
{
#if 0
    uint32_t q[BW];
    for(unsigned int j=0; j<BW; j++)
        q[j] = b[index2(BL-1,j)];

    for(unsigned int i=BL-1; i>0; i--)
        for(unsigned int j=0; j<BW; j++)
            b[index2(i,j)] = b[index2(i-1,j)];
    
    for(unsigned int j=0; j<BW; j++)
        b[index2(0,j)] = q[j];

    
    for(unsigned int i=0; i<12; i++)
        b[index2(i+1,i%BW)] ^= a[i+1];

   
    uint32_t A[MS];
    
    for(unsigned int i=0; i<MS; i++)
        A[i] = a[i]^(a[(i+1)%MS]|(~a[(i+2)%MS]));
   
    for(unsigned int i=0; i<MS; i++)
        a[i] = ROR2(A[(7*i)%MS], i*(i+1)/2);
    
    for(unsigned int i=0; i<MS; i++)
        A[i] = a[i]^a[(i+1)%MS]^a[(i+4)%MS];
   
    A[0] ^= 1;
   
    for(unsigned int i=0; i<MS; i++) 
        a[i] = A[i];

   
    for(unsigned int j=0; j<BW; j++)
        a[j+13] ^= q[j];
#endif 
}

__global__ void InputFunction(uint32_t* in, uint32_t* a, uint32_t* b)
{  
    for(unsigned int j=0; j<BW; j++) 
        a[j+16] ^= in[j];
    
    for(unsigned int j=0; j<BW; j++) 
        b[index2(0,j)] ^= in[j];
}

__global__ void OutputFunction(uint32_t* out, uint32_t* a)
{
    for(unsigned int j=0; j<2; j++)
        out[j] = a[j+1];
}

__global__ void inLoop(uint32_t* in, char* input, uint32_t* p)
{   
    for(unsigned int q=0; q<BW; q++) {
            in[q] = 0;
            for(unsigned int w=0; w<sizeof(uint32_t); w++)
                in[q] |= (uint32_t)((unsigned char)(input[(*p)+q*sizeof(uint32_t)+w])) << (8*w);
    }
    (*p) += sizeof(uint32_t)*BW;
}

__global__ void outLoop(uint32_t* out, char* output, uint32_t* i)
{   
    for(unsigned int q=0; q<2; q++)
            for(unsigned int w=0; w<sizeof(uint32_t); w++)
                output[(*i)*sizeof(uint32_t)*2+q*sizeof(uint32_t)+w] = (char)((out[q] >> (8*w)) & 0xFF);
    (*i)++;
}